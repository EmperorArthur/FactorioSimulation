#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "BeltEntity.cuh"
#include "BlueprintStringReader.cuh"

#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <vector>
#include <time.h>

using namespace std;

#ifdef __unix__ 

void printAndMoveCursorBack(string str)
{
	// not implemented for linux
}

#elif defined(_WIN32) || defined(WIN32)

#include <windows.h>

void printAndMoveCursorBack(string str)
{
	cout << str;
	COORD pos;
	pos.X = 0;
	CONSOLE_SCREEN_BUFFER_INFO nfo;
	GetConsoleScreenBufferInfo(GetStdHandle(STD_OUTPUT_HANDLE), &nfo);
	pos.Y = nfo.dwCursorPosition.Y;
	SetConsoleCursorPosition(GetStdHandle(STD_OUTPUT_HANDLE), pos);
}

#endif

bool useCPU = true;
bool testInputBalance = true;
bool testOuputBalance = true;
bool testFullLoadTroughput = true;
bool testAllTwoBeltTroughputCombinations = false;
int threads = 256;

bool updateEntities(BeltEntity* entities, size_t size, unsigned int iterations)
{
	return useCPU ? updateOnCPU(entities, size, iterations) : updateOnGPU(entities, size, iterations, threads);
}

void displayEntities(BeltEntity* entities, size_t size)
{
	for (unsigned int i = 0; i < size; i++)
	{
		char t = '?';
		switch (entities[i].type)
		{
		case TYPE_BELT:
			t = 'b';
			break;
		case TYPE_LEFT_SPLITTER:
		case TYPE_RIGHT_SPLITTER:
			t = 's';
			break;
		case TYPE_SPAWN:
			t = '+';
			break;
		case TYPE_VOID:
			t = '-';
			break;
		case TYPE_UNDERGROUND_ENTRANCE:
		case TYPE_UNDERGROUND_EXIT:
			t = 'u';
			break;
		case TYPE_BLOCK:
			t = 'X';
			break;
		}

#ifndef _DEBUG
		if (t == '+' || t == '-')
#endif
			cout << "(" << t << ", " << entities[i].buffer << ", " << entities[i].lastTroughput << ", " << i - 1 << ", " << entities[i].next << ")" << endl;
	}
	cout << endl;
}

string loadBlueprintFile(string fileName)
{
	string output;
	ifstream t(fileName);
	stringstream ss;
	ss << t.rdbuf();
	t.close();
	output = ss.str();
	if (output == "")
	{
		ss.clear();
		t.open(getenv("APPDATA") + string("\\factorio\\script-output\\blueprint-string\\") + fileName);
		ss << t.rdbuf();
		t.close();
		output = ss.str();
	}
	return output;
}

struct IdTroughputHelper
{
	int id;
	float troughput;
};

void testBalance(BeltEntity* entities, size_t size, int iterations)
{
	vector<IdTroughputHelper> spawnBelts;
	vector<IdTroughputHelper> voidBelts;

	for (unsigned int i = 0; i < size; i++)
	{
		IdTroughputHelper t;
		t.id = i;
		t.troughput = entities[i].maxTroughput;
		if (entities[i].type == TYPE_SPAWN)
		{
			spawnBelts.push_back(t);
		}
		else if (entities[i].type == TYPE_VOID)
		{
			voidBelts.push_back(t);
		}
	}

	cout << "Testing a " << spawnBelts.size() << " to " << voidBelts.size() << " balancer" << endl;

	BeltEntity* workingCopy = new BeltEntity[size];

	if(testOuputBalance)
	{
		int passedInputs = 0;
		int troughputLimitedInputs = 0;

		for (unsigned int i = 0; i < spawnBelts.size(); i++)
		{
			printAndMoveCursorBack("Progress: " + to_string(passedInputs) + "(" + to_string(i) + ") / " + to_string(spawnBelts.size()));

			memcpy(workingCopy, entities, size * sizeof(BeltEntity));

			for (unsigned int j = 0; j < spawnBelts.size(); j++)
			{
				if (i != j)
				{
					workingCopy[spawnBelts[j].id].maxTroughput = 0;
				}
			}

			updateEntities(workingCopy, size, iterations);

			float expectedResult = workingCopy[voidBelts[0].id].lastTroughput;
			int passedOutputs = 1;

			for (unsigned int j = 1; j < voidBelts.size(); j++)
			{
				float v = workingCopy[voidBelts[j].id].lastTroughput;
				if (fabsf(expectedResult - v) / expectedResult < 0.001)
				{
					passedOutputs++;
				}
				else
				{
#ifdef _DEBUG
					cout << "Output is " << v << " while expected is " << expectedResult << " (" << (fabsf(expectedResult - v) / expectedResult) << ")" << endl;
#endif
				}
			}

			if (workingCopy[spawnBelts[i].id].lastTroughput - workingCopy[spawnBelts[i].id].maxTroughput < -0.001)
			{
				troughputLimitedInputs++;
			}

			if (passedOutputs == voidBelts.size())
			{
				passedInputs++;
			}
		}

		cout << "Output balance: " << passedInputs << "/" << spawnBelts.size();
		if (troughputLimitedInputs > 0)
		{
			cout << "  (" << troughputLimitedInputs << " input" << ((troughputLimitedInputs == 1) ? " is" : "s are") << " troughput limited)" << endl;
		}
		else
		{
			cout << "              " << endl;
		}
	}

	if (testInputBalance)
	{
		int passedOutputs = 0;
		int troughputLimitedOutputs = 0;

		for (unsigned int i = 0; i < voidBelts.size(); i++)
		{
			printAndMoveCursorBack("Progress: " + to_string(passedOutputs) + "(" + to_string(i) + ") / " + to_string(voidBelts.size()));

			memcpy(workingCopy, entities, size * sizeof(BeltEntity));

			for (unsigned int j = 0; j < voidBelts.size(); j++)
			{
				if (i != j)
				{
					workingCopy[voidBelts[j].id].maxTroughput = 0;
				}
			}

			updateEntities(workingCopy, size, iterations);

			float expectedResult = workingCopy[spawnBelts[0].id].lastTroughput;
			int passedInputs = 1;

			for (unsigned int j = 1; j < spawnBelts.size(); j++)
			{
				float s = workingCopy[spawnBelts[j].id].lastTroughput;
				if (fabsf(expectedResult - s) / expectedResult < 0.001)
				{
					passedInputs++;
				}
				else
				{
#ifdef _DEBUG
					cout << "Output is " << s << " while expected is " << expectedResult << " (" << (fabsf(expectedResult - s) / expectedResult) << ")" << endl;
#endif
				}
			}

			if (workingCopy[voidBelts[i].id].lastTroughput - workingCopy[voidBelts[i].id].maxTroughput < -0.001)
			{
				troughputLimitedOutputs++;
			}

			if (passedInputs == spawnBelts.size())
			{
				passedOutputs++;
			}
		}

		cout << "Input balance: " << passedOutputs << "/" << voidBelts.size();
		if (troughputLimitedOutputs > 0)
		{
			cout << "   (" << troughputLimitedOutputs << " output" << ((troughputLimitedOutputs == 1) ? " is" : "s are") << " troughput limited)" << endl;
		}
		else
		{
			cout << "              " << endl;
		}
	}

	if (testFullLoadTroughput)
	{
		memcpy(workingCopy, entities, size * sizeof(BeltEntity));

		updateEntities(workingCopy, size, iterations);

		double maxInput = 0;
		double maxOutput = 0;

		for (unsigned int i = 0; i < size; i++)
		{
			if (entities[i].type == TYPE_SPAWN)
			{
				maxInput += entities[i].spawnAmount;
			}
			else if (entities[i].type == TYPE_VOID)
			{
				maxOutput += entities[i].voidAmount;
			}
		}

		maxOutput = min(maxInput, maxOutput);

		double actualOutput = 0;

		for (unsigned int i = 0; i < size; i++)
		{
			if (workingCopy[i].type == TYPE_VOID)
			{
				actualOutput += workingCopy[i].lastTroughput;
			}
		}

		double troughputPercentage = ((int)(actualOutput / maxOutput * 1000)) / 10.0;

		cout << "Troughput under full load: " << troughputPercentage << "%" << endl;
	}

	if (testAllTwoBeltTroughputCombinations)
	{
		BeltEntity* allBlocked = new BeltEntity[size];
		memcpy(allBlocked, entities, size * sizeof(BeltEntity));
		for (unsigned int i = 0; i < spawnBelts.size(); i++)
		{
			allBlocked[spawnBelts[i].id].spawnAmount = 0;
		}
		for (unsigned int i = 0; i < voidBelts.size(); i++)
		{
			allBlocked[voidBelts[i].id].voidAmount = 0;
		}

		double minTroughput = 100;
		int tested = 0;
		double lastProgress = -1;
		int toTest = ((spawnBelts.size() - 1) * (spawnBelts.size()) / 2) * ((voidBelts.size() - 1) * (voidBelts.size()) / 2);

		for (unsigned int i1 = 0; i1 < spawnBelts.size() - 1; i1++) for (unsigned int i2 = i1 + 1; i2 < spawnBelts.size(); i2++)
		{
			for (unsigned int o1 = 0; o1 < voidBelts.size() - 1; o1++) for (unsigned int o2 = o1 + 1; o2 < voidBelts.size(); o2++)
			{
				double progress = ((int)((tested++ / (double)toTest) * 1000)) / 10.0;
				if (progress != lastProgress)
				{
					stringstream ss;
					ss << "Min troughput: " << minTroughput << "%  Progress: " << progress << ((progress - ((int)progress) == 0) ? ".0%   " : "%   ");
					printAndMoveCursorBack(ss.str());
					lastProgress = progress;
				}

				memcpy(workingCopy, allBlocked, size * sizeof(BeltEntity));
				workingCopy[spawnBelts[i1].id].spawnAmount = spawnBelts[i1].troughput;
				workingCopy[spawnBelts[i2].id].spawnAmount = spawnBelts[i2].troughput;
				workingCopy[voidBelts[o1].id].voidAmount = voidBelts[o1].troughput;
				workingCopy[voidBelts[o2].id].voidAmount = voidBelts[o2].troughput;

				updateEntities(workingCopy, size, iterations);

				double maxInput = 0;
				double maxOutput = 0;

				maxInput += spawnBelts[i1].troughput;
				maxInput += spawnBelts[i2].troughput;
				maxOutput += voidBelts[o1].troughput;
				maxOutput += voidBelts[o2].troughput;

				maxOutput = min(maxInput, maxOutput);

				double actualOutput = 0;

				actualOutput += workingCopy[voidBelts[o1].id].lastTroughput;
				actualOutput += workingCopy[voidBelts[o2].id].lastTroughput;

				double troughputPercentage = ((int)(actualOutput / maxOutput * 1000)) / 10.0;

				if (troughputPercentage < minTroughput)
				{
					minTroughput = troughputPercentage;
				}
			}
		}

		cout << "Min troughput with two belts: " << minTroughput << "%                          " << endl;
	}

	cout << endl;
	delete[] workingCopy;
}

void printHelp()
{
	cout << "beltbalancer.exe -f=YOUR_BALANCER_FILE.txt ([-cpu]|[-gpu]|[-cudadev=N]) [-t] [-i=N] [-benchmark] [-o]" << endl;
}

int main(int argc, char** argv)
{
	hipError_t cudaStatus;

	int iterations = -1;
	string file = "DUMB_ASS";
	int cudaDeviceId = 0;
	bool timeIt = false;
	bool optimize = false;

	for (int i = 1; i < argc; i++)
	{
		string arg = argv[i];
		if (arg.compare(0, 3, "-f=") == 0)
		{
			file = arg.substr(3, arg.length() - 3);
		}
		else if (arg.compare("-o") == 0)
		{
			optimize = true;
		}
		else if (arg.compare("-t") == 0)
		{
			testAllTwoBeltTroughputCombinations = true;
		}
		else if (arg.compare("-cpu") == 0)
		{
			useCPU = true;
		}
		else if (arg.compare("-gpu") == 0)
		{
			useCPU = false;
		}
		else if (arg.compare(0, 9, "-threads=") == 0)
		{
			threads = stoi(arg.substr(9));
			useCPU = false;
		}
		else if (arg.compare(0, 9, "-cudadev=") == 0)
		{
			cudaDeviceId = stoi(arg.substr(9));
			useCPU = false;
		}
		else if (arg.compare(0, 3, "-i=") == 0)
		{
			iterations = stoi(arg.substr(3));
		}
		else if (arg.compare("-benchmark") == 0)
		{
			timeIt = true;
		}
		else if (arg.compare("-h") == 0 || arg.compare("-?"))
		{
			printHelp();
			return 0;
		}
	}

	if (file.compare("DUMB_ASS") == 0)
	{
		printHelp();
		return 0;
	}

	if (!useCPU)
	{
		// Choose which GPU to run on
		cudaStatus = hipSetDevice(cudaDeviceId);
		if (cudaStatus != hipSuccess)
		{
			cerr << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?" << endl;
			return 1;
		}
	}

	size_t size = 0;
	BeltEntity* belts = parseBlueprintString(loadBlueprintFile(file), &size, optimize);

	if (iterations == -1)
	{
		iterations = size * 5;
	}

	if (size == 0)
	{
		cerr << "Loading blueprint failed" << endl;
		return 1;
	}

	if (!timeIt)
	{
		testBalance(belts, size, iterations);
	}
	else
	{
		clock_t start;
		clock_t end;

		start = clock();

		updateEntities(belts, size, iterations);

		end = clock();

		double timeTaken = (end - start) / (double)CLOCKS_PER_SEC;

		cout << "Simulating " << size << " belt parts for " << iterations << " iterations took " << timeTaken << " seconds." << endl << endl;
	}

	delete[] belts;

	if (!useCPU)
	{
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}
	}

    return 0;
}
