#include "hip/hip_runtime.h"
#include "BeltEntity.cuh"
#include <vector>
#include <iostream>
#include <thread>

#define MIN(x, y) ((x) < (y) ? x : y)

using namespace std;

#ifdef _DEBUG
#define DEBUG(x) (x)
#else
#define DEBUG(x) ;
#endif

__device__ inline void updatePass1(BeltEntity* entities, int i)
{
	BeltEntity* b = entities + i;

	switch (b->type)
	{
	case TYPE_SPAWN:
		b->buffer = b->spawnAmount;
	case TYPE_BELT:
	case TYPE_UNDERGROUND_ENTRANCE:
	case TYPE_UNDERGROUND_EXIT:
		BeltEntity* next = entities + b->next;
		next->addToBuffer = MIN(b->maxThroughput, b->buffer);
		if (next->addToBuffer + next->buffer > next->maxThroughput * 2)
		{
			next->addToBuffer = next->maxThroughput * 2 - next->buffer;
		}
		b->subtractFromBuffer = next->addToBuffer;
		break;
	case TYPE_VOID:
		b->subtractFromBuffer = MIN(b->buffer, b->voidAmount);
		break;
	case TYPE_LEFT_SPLITTER:
		BeltEntity* r = entities + b->otherSplitterPart;
		BeltEntity* lnext = entities + b->next;
		BeltEntity* rnext = entities + r->next;
		float ldemand = lnext->maxThroughput * 2 - lnext->buffer;
		ldemand = MIN(ldemand, lnext->maxThroughput);
		ldemand = MIN(ldemand, b->maxThroughput);
		float rdemand = rnext->maxThroughput * 2 - rnext->buffer;
		rdemand = MIN(rdemand, rnext->maxThroughput);
		rdemand = MIN(rdemand, r->maxThroughput);
		float lsupply = MIN(b->maxThroughput, b->buffer);
		float rsupply = MIN(r->maxThroughput, r->buffer);
		float demand = ldemand + rdemand;
		float supply = lsupply + rsupply;
		if (demand >= supply)
		{
			float halfSupply = supply / 2;
			if (ldemand < halfSupply)
			{
				lnext->addToBuffer = ldemand;
				rnext->addToBuffer = supply - ldemand;
			}
			else if (rdemand < halfSupply)
			{
				rnext->addToBuffer = rdemand;
				lnext->addToBuffer = supply - rdemand;
			}
			else
			{
				lnext->addToBuffer = halfSupply;
				rnext->addToBuffer = halfSupply;
			}
			b->subtractFromBuffer = lsupply;
			r->subtractFromBuffer = rsupply;
		}
		else
		{
			float halfDemand = demand / 2;
			lnext->addToBuffer = ldemand;
			rnext->addToBuffer = rdemand;
			if (lsupply < halfDemand)
			{
				b->subtractFromBuffer = lsupply;
				r->subtractFromBuffer = demand - lsupply;
			}
			else if (rsupply < halfDemand)
			{
				r->subtractFromBuffer = rsupply;
				b->subtractFromBuffer = demand - rsupply;
			}
			else
			{
				r->subtractFromBuffer = halfDemand;
				b->subtractFromBuffer = halfDemand;
			}
		}
		break;
	case TYPE_RIGHT_SPLITTER: // right splitter part gets updated together with the left part
	case TYPE_BLOCK:
	default:
		break;
	}
}

__device__ inline void updatePass2(BeltEntity* entities, int i)
{
	BeltEntity* b = entities + i;

	b->buffer += b->addToBuffer - b->subtractFromBuffer;
}

__global__ void updateKernel(BeltEntity* entities)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	updatePass1(entities, i);

	__syncthreads();

	updatePass2(entities, i);

	__syncthreads();
}

__global__ void testThroughputKernel(BeltEntity* allEntities, size_t size, unsigned int iterations, int* beltIds, int inputBeltCount,
									 int outputBeltCount, int testCaseCount, float* testCaseData, float* results)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= testCaseCount)
	{
		return;
	}

	BeltEntity* entities = allEntities + index * size + 1;

	int ic = inputBeltCount;
	int oc = outputBeltCount;

	int testCaseOffset = index * (ic + oc);
	int* inputBeltIds = beltIds + testCaseOffset;
	int* outputBeltIds = inputBeltIds + ic;

	for (int i = 0; i < ic; i++)
	{
		entities[inputBeltIds[i]].maxThroughput *= testCaseData[testCaseOffset + i];
	}

	for (int i = 0; i < oc; i++)
	{
		entities[outputBeltIds[i]].maxThroughput *= testCaseData[testCaseOffset + ic + i];
	}

	for (int i = 0; i < iterations; i++)
	{
		for (int j = 0; j < size - 1; j++)
		{
			updatePass1(entities, j);
		}
		for (int j = 0; j < size - 1; j++)
		{
			updatePass2(entities, j);
		}
	}

	float maxInput = 0;
	float maxOutput = 0;

	for (int i = 0; i < ic; i++)
	{
		maxInput += entities[inputBeltIds[i]].maxThroughput;
	}

	for (int i = 0; i < oc; i++)
	{
		maxOutput += entities[outputBeltIds[i]].maxThroughput;
	}

	maxOutput = MIN(maxInput, maxOutput);

	float actualOutput = 0;

	for (int i = 0; i < oc; i++)
	{
		actualOutput += entities[outputBeltIds[i]].lastThroughput;
	}

	results[index] = actualOutput / maxOutput;
}

int countSetBits(unsigned int v)
{
	v = v - ((v >> 1) & 0x55555555);
	v = (v & 0x33333333) + ((v >> 2) & 0x33333333);
	return ((v + (v >> 4) & 0xF0F0F0F) * 0x1010101) >> 24;
}

double testThroughputCombinationsOnGPU(BeltEntity* entities, size_t size, unsigned int iterations, int minPopCount, int maxPopCount)
{
	vector<int> inputIds;
	vector<int> outputIds;

	for (int i = 0; i < size; i++)
	{
		if (entities[i].type == TYPE_SPAWN)
		{
			inputIds.push_back(i - 1);
		}
		else if (entities[i].type == TYPE_VOID)
		{
			outputIds.push_back(i - 1);
		}
	}

	DEBUG(cout << "Detected " << inputIds.size() << " inputs and " << outputIds.size() << " outputs" << endl);

	int testCaseCount = 0;
	int inputBeltCount = inputIds.size();
	int outputBeltCount = outputIds.size();

	vector<float> outputCombinations;
	for (int out = 0; out < (1 << outputBeltCount); out++)
	{
		int popCount = countSetBits(out);
		if (popCount < minPopCount || popCount > maxPopCount)
		{
			continue;
		}
		int outCopy = out;
		for (int i = 0; i < outputBeltCount; i++)
		{
			outputCombinations.push_back((float)(outCopy & 1));
			outCopy = outCopy >> 1;
		}
	}
	int outputCombinationsSize = outputCombinations.size() / outputBeltCount;

	vector<float> testCaseDataVector;
	float* inputData = new float[inputBeltCount];
	for (int in = 0; in < (1 << inputBeltCount); in++)
	{
		int popCount = countSetBits(in);
		if (popCount < minPopCount || popCount > maxPopCount)
		{
			continue;
		}
		int inCopy = in;
		for (int i = 0; i < inputBeltCount; i++)
		{
			inputData[i] = (float)(inCopy & 1);
			inCopy = inCopy >> 1;
		}
		for (int out = 0; out < outputCombinationsSize; out++)
		{
			for (int i = 0; i < inputBeltCount; i++)
			{
				testCaseDataVector.push_back(inputData[i]);
			}
			for (int i = 0; i < outputBeltCount; i++)
			{
				testCaseDataVector.push_back(outputCombinations[out * outputBeltCount + i]);
			}
			testCaseCount++;
		}
	}

	DEBUG(cout << "testCaseCount " << testCaseCount << endl);
	DEBUG(cout << "testCaseDataSize " << (sizeof(float)* testCaseDataVector.size()) << endl);

	float* dev_testCaseData = 0;
	hipMalloc((void**)&dev_testCaseData, sizeof(float)* testCaseDataVector.size());
	hipMemcpy(dev_testCaseData, &testCaseDataVector[0], sizeof(float)*testCaseDataVector.size(), hipMemcpyHostToDevice);

	float* results = (float*)malloc(sizeof(float)* testCaseCount);
	for (unsigned int i = 0; i < testCaseCount; i++)
	{
		results[i] = -0.69f;
	}
	float* dev_results = 0;
	hipMalloc((void**)&dev_results, sizeof(float)* testCaseCount);
	hipMemcpy(dev_results, results, sizeof(float)* testCaseCount, hipMemcpyHostToDevice);

	int* dev_beltIds = 0;
	hipError_t hipError_t = hipMalloc((void**)&dev_beltIds, sizeof(int)* testCaseCount * (inputBeltCount + outputBeltCount));
	if (hipError_t != hipSuccess) {
		cerr << "hipMalloc dev_beltIds failed: " << hipError_t << endl;
	}
	hipMemcpy(dev_beltIds, &inputIds[0], sizeof(int)* inputBeltCount, hipMemcpyHostToDevice);
	hipMemcpy(dev_beltIds + inputBeltCount, &outputIds[0], sizeof(int)* outputBeltCount, hipMemcpyHostToDevice);

	BeltEntity* dev_entities = 0;
	DEBUG(cout << "hipMalloc dev_entities with " << (sizeof(BeltEntity)* size * testCaseCount / 1024) << " kbytes" << endl);
	hipError_t = hipMalloc((void**)&dev_entities, sizeof(BeltEntity)* size * testCaseCount);
	if (hipError_t != hipSuccess) {
		cerr << "hipMalloc entities failed: " << hipError_t << endl;
	}
	hipMemcpy(dev_entities, entities, sizeof(BeltEntity) * size, hipMemcpyHostToDevice);

	// duplicate arrays for the belt ids and the belt structures themselves
	for (int i = 1; i < testCaseCount; i++)
	{
		hipMemcpy(dev_entities + size * i, dev_entities, sizeof(BeltEntity)* size, hipMemcpyDeviceToDevice);
		hipMemcpy(dev_beltIds + (inputBeltCount + outputBeltCount) * i, dev_beltIds, sizeof(int)* (inputBeltCount + outputBeltCount), hipMemcpyDeviceToDevice);
	}

	const int threads = 256;
	int blocks = (testCaseCount - 1) / threads + 1;

	DEBUG(cout << "Launching kernel with dimensions: (" << blocks << "," << threads << ")" << endl);

	testThroughputKernel << <blocks, threads >> >(dev_entities, size, iterations, dev_beltIds, inputBeltCount, outputBeltCount, testCaseCount, dev_testCaseData, dev_results);
	hipDeviceSynchronize();

	DEBUG(cout << "last cuda error: " << hipGetErrorString(hipPeekAtLastError()) << endl);
	
	hipMemcpy(results, dev_results, sizeof(float)* testCaseCount, hipMemcpyDeviceToHost);
	DEBUG(cout << "Copied results to host memory" << endl);

	float minimum = 1;
	for (unsigned int i = 0; i < testCaseCount; i++)
	{
#ifdef _DEBUG
		if (i < 32) cout << "results[" << i << "] = " << results[i] << endl;
#endif
		if (results[i] < minimum)
		{
			minimum = results[i];
		}
	}

	DEBUG(cout << "Free data" << endl);
	free(results);
	hipFree(dev_beltIds);
	hipFree(dev_entities);
	hipFree(dev_testCaseData);
	hipFree(dev_results);

	DEBUG(cout << "Return minimum of " << minimum << endl);
	return minimum;
}

void testThroughput(BeltEntity* source, size_t size, unsigned int iterations, vector<int>& inputIds, vector<int>& outputIds,
					  int startIndex, int endIndex, vector<float>& inputData, vector<float>& outputData, float* results)
{
	BeltEntity* entities = new BeltEntity[size];

	for (int index = startIndex; index < endIndex; index++)
	{
		memcpy(entities, source, size * sizeof(BeltEntity));

		int inputDataSize = inputData.size() / inputIds.size();
		
		int inputOffset = (index % inputDataSize) * inputIds.size();
		int outputOffset = (index / inputDataSize) * outputIds.size();

		for (int i = 0; i < inputIds.size(); i++)
		{
			entities[inputIds[i]].maxThroughput *= inputData[inputOffset + i];
		}

		for (int i = 0; i < outputIds.size(); i++)
		{
			entities[outputIds[i]].maxThroughput *= outputData[outputOffset + i];
		}

		updateOnCPU(entities, size, iterations);

		float maxInput = 0;
		float maxOutput = 0;

		for (int i = 0; i < inputIds.size(); i++)
		{
			maxInput += entities[inputIds[i]].maxThroughput;
		}

		for (int i = 0; i < outputIds.size(); i++)
		{
			maxOutput += entities[outputIds[i]].maxThroughput;
		}

		maxOutput = MIN(maxInput, maxOutput);

		float actualOutput = 0;

		for (int i = 0; i < outputIds.size(); i++)
		{
			actualOutput += entities[outputIds[i]].lastThroughput;
		}

		results[index] = actualOutput / maxOutput;
	}

	delete [] entities;
}

double testThroughputCombinationsOnCPU(BeltEntity* entities, size_t size, unsigned int iterations, int minPopCount, int maxPopCount, int threadCount)
{
	vector<int> inputIds;
	vector<int> outputIds;
	for (int i = 0; i < size; i++)
	{
		if (entities[i].type == TYPE_SPAWN)
		{
			inputIds.push_back(i);
		}
		else if (entities[i].type == TYPE_VOID)
		{
			outputIds.push_back(i);
		}
	}
	
	int inputBeltCount = inputIds.size();
	int outputBeltCount = outputIds.size();

	vector<float> inputCombinations;
	for (int in = 0; in < (1 << inputBeltCount); in++)
	{
		int popCount = countSetBits(in);
		if (popCount < minPopCount || popCount > maxPopCount)
		{
			continue;
		}
		int inCopy = in;
		for (int i = 0; i < inputBeltCount; i++)
		{
			inputCombinations.push_back((float)(inCopy & 1));
			inCopy = inCopy >> 1;
		}
	}
	int inputCombinationsSize = inputCombinations.size() / inputBeltCount;

	vector<float> outputCombinations;
	for (int out = 0; out < (1 << outputBeltCount); out++)
	{
		int popCount = countSetBits(out);
		if (popCount < minPopCount || popCount > maxPopCount)
		{
			continue;
		}
		int outCopy = out;
		for (int i = 0; i < outputBeltCount; i++)
		{
			outputCombinations.push_back((float)(outCopy & 1));
			outCopy = outCopy >> 1;
		}
	}
	int outputCombinationsSize = outputCombinations.size() / outputBeltCount;

	int testCases = outputCombinationsSize * inputCombinationsSize;
	
	vector<float> result(testCases, 69.0f);
	
	threadCount = MIN(threadCount, result.size());
	
	if (threadCount <= 1)
	{
		testThroughput(entities, size, iterations, inputIds, outputIds, 0, result.size(), inputCombinations, outputCombinations, &result[0]);
	}
	else
	{
		thread** threads = new thread*[threadCount];
		
		for (int i = 0; i < threadCount; i++)
		{
			int startIndex = (result.size() / threadCount) * i;
			int endIndex = (result.size() / threadCount) * (i + 1);
			if (i == threadCount - 1)
			{
				endIndex = result.size();
			}
			threads[i] = new thread(testThroughput, entities, size, iterations, inputIds, outputIds, startIndex, endIndex, inputCombinations, outputCombinations, &result[0]);
		}
		
		for (int i = 0; i < threadCount; i++)
		{
			threads[i]->join();
			delete threads[i];
		}
		
		delete[] threads;
	}

	double minimum = 69;
	
	for (int i = 0; i < result.size(); i++)
	{
		if (result[i] < minimum)
		{
			minimum = result[i];
		}
	}
	
	return minimum;
}

bool updateOnGPU(BeltEntity* entities, size_t size, unsigned int iterations, int threads)
{
	BeltEntity* dev_entities = 0;
	hipError_t cudaStatus;

	int nSize = size;
	int blocks = (nSize - 1) / threads + 1;
	int fSize = blocks * threads;

	BeltEntity* paddingBlocks = new BeltEntity[fSize - nSize];

	for (int i = 0; i < (fSize - nSize); i++)
	{
		BeltEntity b;
		b.type = TYPE_BLOCK;
		b.maxThroughput = 0;
		b.addToBuffer = 0;
		b.buffer = 0;
		b.subtractFromBuffer = 0;
		b.next = -1;
		b.otherSplitterPart = -1;
		paddingBlocks[i] = b;
	}

	cudaStatus = hipMalloc((void**)&dev_entities, (fSize)* sizeof(BeltEntity));
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_entities, entities, (nSize)* sizeof(BeltEntity), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_entities + nSize, paddingBlocks, (fSize - nSize)* sizeof(BeltEntity), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	for (unsigned int i = 0; i < iterations; i++)
	{
		updateKernel << <blocks, threads >> >(dev_entities + 1);
	}

	cudaStatus = hipMemcpy(entities, dev_entities, (nSize) * sizeof(BeltEntity), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

Error:
	hipFree(dev_entities);

	return cudaStatus == hipSuccess;
}

bool updateOnCPU(BeltEntity* entities, size_t size, unsigned int iterations)
{
	for (unsigned int j = 0; j < iterations; j++)
	{
		for (int i = 1; i < size; i++)
		{
			BeltEntity* b = entities + i;
			float ldemand = 0;
			float rdemand = 0;
			float lsupply = 0;
			float rsupply = 0;
			float demand = 0;
			float supply = 0;
			BeltEntity* r = 0;
			BeltEntity* lnext = 0;
			BeltEntity* rnext = 0;
			BeltEntity* next = 0;

			switch (b->type)
			{
			case TYPE_SPAWN:
				b->buffer = b->spawnAmount;
			case TYPE_BELT:
			case TYPE_UNDERGROUND_ENTRANCE:
			case TYPE_UNDERGROUND_EXIT:
				next = entities + b->next + 1;
				next->addToBuffer = MIN(b->maxThroughput, b->buffer);
				if (next->addToBuffer + next->buffer > next->maxThroughput * 2)
				{
					next->addToBuffer = next->maxThroughput * 2 - next->buffer;
				}
				b->subtractFromBuffer = next->addToBuffer;
				break;
			case TYPE_VOID:
				b->subtractFromBuffer = MIN(b->buffer, b->voidAmount);
				break;
			case TYPE_LEFT_SPLITTER:
				r = entities + b->otherSplitterPart + 1;
				lnext = entities + b->next + 1;
				rnext = entities + r->next + 1;
				ldemand = lnext->maxThroughput * 2 - lnext->buffer;
				ldemand = MIN(ldemand, lnext->maxThroughput);
				ldemand = MIN(ldemand, b->maxThroughput);
				rdemand = rnext->maxThroughput * 2 - rnext->buffer;
				rdemand = MIN(rdemand, rnext->maxThroughput);
				rdemand = MIN(rdemand, r->maxThroughput);
				lsupply = MIN(b->maxThroughput, b->buffer);
				rsupply = MIN(r->maxThroughput, r->buffer);
				demand = ldemand + rdemand;
				supply = lsupply + rsupply;
				if (demand >= supply)
				{
					float halfSupply = supply / 2;
					if (ldemand < halfSupply)
					{
						lnext->addToBuffer = ldemand;
						rnext->addToBuffer = supply - ldemand;
					}
					else if (rdemand < halfSupply)
					{
						rnext->addToBuffer = rdemand;
						lnext->addToBuffer = supply - rdemand;
					}
					else
					{
						lnext->addToBuffer = halfSupply;
						rnext->addToBuffer = halfSupply;
					}
					b->subtractFromBuffer = lsupply;
					r->subtractFromBuffer = rsupply;
				}
				else
				{
					float halfDemand = demand / 2;
					lnext->addToBuffer = ldemand;
					rnext->addToBuffer = rdemand;
					if (lsupply < halfDemand)
					{
						b->subtractFromBuffer = lsupply;
						r->subtractFromBuffer = demand - lsupply;
					}
					else if (rsupply < halfDemand)
					{
						r->subtractFromBuffer = rsupply;
						b->subtractFromBuffer = demand - rsupply;
					}
					else
					{
						r->subtractFromBuffer = halfDemand;
						b->subtractFromBuffer = halfDemand;
					}
				}
				break;
			case TYPE_RIGHT_SPLITTER: // right splitter part gets updated together with the left part
			default:
				break;
			}
		}

		for (int i = 1; i < size; i++)
		{
			BeltEntity* b = entities + i;
			b->buffer += b->addToBuffer - b->subtractFromBuffer;
		}
	}

	return true;
}
